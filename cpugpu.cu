#include "hip/hip_runtime.h"
#include <iostream>
#include <hip/hip_runtime.h>
#include <stdlib.h> // for strtol
#include "Graph.h"
#include <omp.h>

#define MAX_THREAD_COUNT 1024
#define CEIL(a, b) ((a - 1) / b + 1)

#define catchCudaError(error) { gpuAssert((error), __FILE__, __LINE__); }

using namespace std;

float device_time_taken;

void printTime(float ms) {
  printf("%d,", (int)ms);
}

// Catch Cuda errors
inline void gpuAssert(hipError_t error, const char *file, int line, bool abort = false) {
    if (error != hipSuccess) {
        printf("\nCUDA error code %i: %s\n", error, hipGetErrorString(error));
        printf("\nIn file: %s on line: %d\n", file, line);

        if (abort)
            exit(-1);
    }
}

__global__ void betweennessCentralityKernel(Graph *graph, float *bwCentrality, 
                                            int nodeFrom, int nodeTo, int nodeCount,
                                            int *sigma, int *distance, float *dependency)
{
    int idx = threadIdx.x;
    if(idx >= max((2*(graph->edgeCount)), nodeCount))
        return;

    __shared__ int s;
    __shared__ int current_depth;
    __shared__ bool done;

    if(idx == 0) {
        s = nodeFrom - 1;
    }
    __syncthreads();

    while(s <= nodeTo) {
        if (idx == 0) {
            ++s;
            done = false;
            current_depth = -1;
        }
        __syncthreads();
    
        for (int i = idx; i < nodeCount; i += blockDim.x) {
            if (i == s) {
                distance[i] = 0;
                sigma[i] = 1;
            } else {
                distance[i] = INT_MAX;
                sigma[i] = 0;
            }
            dependency[i]= 0.0;
        }
        __syncthreads();

        while (!done) {
            __syncthreads();

            if (threadIdx.x == 0) {
                current_depth++;
            }
            done = true;
            __syncthreads();

            for (int i = idx; i < (2*(graph->edgeCount)); i += blockDim.x) {
                int v = graph->edgeList1[i];
                if (distance[v] == current_depth) {
                    int w = graph->edgeList2[i];
                    if (distance[w] == INT_MAX) {
                        distance[w] = distance[v] + 1;
                        done = false;
                    }
                    if (distance[w] == (distance[v] + 1)) {
                        atomicAdd(&sigma[w], sigma[v]);
                    }
                }
            }
            __syncthreads();
        }
        __syncthreads();

        // Reverse BFS
        while (current_depth) {
            if (idx == 0){
                current_depth--;
            }
            __syncthreads();

            for (int i = idx; i < (2*(graph->edgeCount)); i += blockDim.x)  {
                int v = graph->edgeList1[i];
                if(distance[v] == current_depth) {
                    int w = graph->edgeList2[i];
                    if(distance[w] == (distance[v] + 1)) {
                        if (sigma[w] != 0) {
                            atomicAdd(dependency + v, (sigma[v] * 1.0 / sigma[w]) * (1 + dependency[w]));
                        }
                    }
                }
            }
            __syncthreads();
        }

        for (int v = idx; v < nodeCount; v += blockDim.x) {
            if (v != s) {
                bwCentrality[v] += dependency[v] / 2;
            }
        }
        __syncthreads();
    }
}

float *betweennessCentralityCPU(Graph *graph, int nodeFrom, int nodeTo) {
    const int nodeCount = graph->getNodeCount();
    const int edgeCount = graph->getEdgeCount();

    float *bcs = new float[nodeCount]();

    for (int i = nodeFrom; i < nodeTo; ++i) {
      bcs[i] = 0.0;
    }

    #pragma omp parallel
    {
      vector<int> adjacencyListPointers(
        graph->adjacencyListPointers,
        graph->adjacencyListPointers + nodeCount + 1);
      vector<int> adjacencyList(graph->adjacencyList,
                                graph->adjacencyList + 2 * edgeCount + 1);
      vector<double> dependency(nodeCount, 0);
      vector<int> sigma(nodeCount, 0);
      vector<int> distance(nodeCount, -1);
      vector<vector<int> > predecessor(nodeCount);
      vector<double> localBcs(nodeCount, 0.0);
      stack<int> st;
      queue<int> q;

      for (int i = nodeFrom; i < nodeTo; ++i) {
        predecessor[i].reserve(20);
      }

      #pragma omp for schedule(dynamic, 4)
      for (int s = nodeFrom; s <= nodeTo; s++) {
        printf( "Thread %d works with s_node %d\n", omp_get_thread_num(), s);
        for (int i = nodeFrom; i < nodeTo; ++i) {
            predecessor[i].clear();
        }

        fill(dependency.begin(), dependency.end(), 0);
        fill(sigma.begin(), sigma.end(), 0);
        fill(distance.begin(), distance.end(), -1);

        distance[s] = 0;
        sigma[s] = 1;
        q.push(s);
        while (!q.empty()) {
          int v = q.front();
          q.pop();
          st.push(v);

          for (int i = graph->adjacencyListPointers[v]; i < graph->adjacencyListPointers[v + 1]; i++) {
            int w = graph->adjacencyList[i];
            if (distance[w] < 0) {
              q.push(w);
              distance[w] = distance[v] + 1;
            }
            if (distance[w] == distance[v] + 1) {
              sigma[w] += sigma[v];
              predecessor[w].push_back(v);
            }
          }
        }

        while (!st.empty()) {
          int w = st.top();
          st.pop();

          for (const int &v : predecessor[w]) {
            if (sigma[w] != 0)
              dependency[v] += (sigma[v] * 1.0 / sigma[w]) * (1 + dependency[w]);
          }
          if (w != s) {
            localBcs[w] += dependency[w] / 2;
          }
        }

        #pragma omp critical
        {
          for (int i = 0; i < nodeCount; ++i) {
            bcs[i] += localBcs[i];
          }
        }
      }
    }
    cout << endl;
    return bcs;
}

float *betweennessCentrality(Graph *graph, int nodeCount, int nodeFrom, int nodeTo)
{
    float *bwCentrality = new float[nodeCount]();
    float *device_bwCentrality, *dependency;
    int *sigma, *distance;

    //TODO: Allocate device memory for bwCentrality
    catchCudaError(hipMalloc((void **)&device_bwCentrality, sizeof(float) * nodeCount));
    catchCudaError(hipMalloc((void **)&sigma, sizeof(int) * nodeCount));
    catchCudaError(hipMalloc((void **)&distance, sizeof(int) * nodeCount));
    catchCudaError(hipMalloc((void **)&dependency, sizeof(float) * nodeCount));
    catchCudaError(hipMemcpy(device_bwCentrality, bwCentrality, sizeof(float) * nodeCount, hipMemcpyHostToDevice));

    // Timer
    hipEvent_t device_start, device_end;
    catchCudaError(hipEventCreate(&device_start));
    catchCudaError(hipEventCreate(&device_end));
    catchCudaError(hipEventRecord(device_start));

    betweennessCentralityKernel<<<1, MAX_THREAD_COUNT>>>(
      graph, 
      device_bwCentrality, 
      nodeFrom,
      nodeTo,
      nodeCount, 
      sigma, 
      distance, 
      dependency
    );

    hipDeviceSynchronize();
    
    //End of progress bar
    // cout << endl;

    // Timer
    catchCudaError(hipEventRecord(device_end));
    catchCudaError(hipEventSynchronize(device_end));
    hipEventElapsedTime(&device_time_taken, device_start, device_end);

    // Copy back and free memory
    catchCudaError(hipMemcpy(bwCentrality, device_bwCentrality, sizeof(float) * nodeCount, hipMemcpyDeviceToHost));
    catchCudaError(hipFree(device_bwCentrality));
    catchCudaError(hipFree(sigma));
    catchCudaError(hipFree(dependency));
    catchCudaError(hipFree(distance));

    return bwCentrality;
}

int main(int argc, char *argv[]) {
    if (argc < 2) {
        cout << "Usage: " << argv[0] << " <input_file> <output_file>\n";
        return 0;
    }

    freopen(argv[1], "r", stdin);

    const long threshold_percent = strtol(argv[3], NULL, 10);

    Graph *host_graph = new Graph();
    Graph *device_graph;

    catchCudaError(hipMalloc((void **)&device_graph, sizeof(Graph)));
    host_graph->readGraph();
    host_graph->convertToCOO();

    int nodeCount = host_graph->getNodeCount();
    int edgeCount = host_graph->getEdgeCount();
    catchCudaError(hipMemcpy(device_graph, host_graph, sizeof(Graph), hipMemcpyHostToDevice));

    // Copy edge List to device
    int *edgeList1;
    int *edgeList2;

    // Alocate device memory and copy
    catchCudaError(hipMalloc((void **)&edgeList1, sizeof(int) * (2 * edgeCount + 1)));
    catchCudaError(hipMemcpy(edgeList1, host_graph->edgeList1, sizeof(int) * (2 * edgeCount + 1), hipMemcpyHostToDevice));

    catchCudaError(hipMalloc((void **)&edgeList2, sizeof(int) * (2 * edgeCount + 1)));
    catchCudaError(hipMemcpy(edgeList2, host_graph->edgeList2, sizeof(int) * (2 * edgeCount + 1), hipMemcpyHostToDevice));

    // Update the pointer to this, in device_graph
    catchCudaError(hipMemcpy(&(device_graph->edgeList1), &edgeList1, sizeof(int *), hipMemcpyHostToDevice));
    catchCudaError(hipMemcpy(&(device_graph->edgeList2), &edgeList2, sizeof(int *), hipMemcpyHostToDevice));

    const int threshold = (int) ((float)nodeCount * (float)threshold_percent / (float) 100);
    float *bwCentralityGPU;
    float *bwCentralityCPU;

    clock_t start, end;

    #pragma omp parallel sections
    {
      #pragma omp section
      {
        const int nodeFrom = threshold;
        const int nodeTo = nodeCount - 1;
        bwCentralityGPU = betweennessCentrality(device_graph, nodeCount, nodeFrom, nodeTo);
      }

      #pragma omp section
      {
        const int nodeFrom = 0;
        const int nodeTo = threshold - 1;
        
        start = clock();
        bwCentralityCPU = betweennessCentralityCPU(host_graph, nodeFrom, nodeTo);
      }
    }

    float *bwCentrality = new float[nodeCount];
    float maxBetweenness = -1;
    for (int i = 0; i < nodeCount; i++) {
      bwCentrality[i] = bwCentralityCPU[i] + bwCentralityGPU[i];
      maxBetweenness = max(maxBetweenness, bwCentrality[i]);
    }
    end = clock();

    float host_time_taken = 1000.0 * (end - start) / (float)CLOCKS_PER_SEC;

    printf("%s, %s, ", argv[1], argv[3]);
    printf("%0.2lf, ", maxBetweenness);
    printTime(device_time_taken);
    printTime(host_time_taken);
    printTime(max(device_time_taken, host_time_taken));

    if (argc == 3) {
      freopen(argv[2], "w", stdout);
      for (int i = 0; i < nodeCount; i++)
        cout << bwCentrality[i] << " ";
      cout << endl;
    }

    // Free all memory
    delete[] bwCentrality;
    catchCudaError(hipFree(edgeList1));
    catchCudaError(hipFree(edgeList2));
    catchCudaError(hipFree(device_graph));
}
