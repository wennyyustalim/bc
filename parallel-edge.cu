#include "hip/hip_runtime.h"
#include <iostream>
#include <hip/hip_runtime.h>
#include "Graph.h"

#define MAX_THREAD_COUNT 1024
#define CEIL(a, b) ((a - 1) / b + 1)

using namespace std;

#define catchCudaError(error) { gpuAssert((error), __FILE__, __LINE__); }

float device_time_taken;

void printTime(float ms) {
  // int h = ms / (1000 * 3600);
  // int m = (((int)ms) / (1000 * 60)) % 60;
  // int s = (((int)ms) / 1000) % 60;
  // int intMS = ms;
  // intMS %= 1000;
  // printf("Time Taken (Parallel) = %dh %dm %ds %dms\n", h, m, s, intMS);
  printf("%d,", (int)ms);
}

inline void gpuAssert(hipError_t error, const char *file, int line, bool abort = false) {
  if (error != hipSuccess) {
    printf("\n====== Cuda Error Code %i ======\n %s", error, hipGetErrorString(error));
    printf("\nIn file :%s\nOn line: %d", file, line);

    if (abort)
      exit(-1);
  }
}

__global__ void betweennessCentralityKernel(Graph *graph, float *bwCentrality, 
                                            int nodeFrom, int nodeTo, int nodeCount,
                                            int *sigma, int *distance, float *dependency) {
  int idx = threadIdx.x;
  if (idx >= max((2*(graph->edgeCount)), nodeCount))
    return;

  __shared__ int s;
  __shared__ int current_depth;
  __shared__ bool done;

  if (idx == 0) {
    s = nodeFrom - 1;
  }
  __syncthreads();

  while (s <= nodeTo) {
    if (idx == 0) {
        ++s;
        done = false;
        current_depth = -1;
    }
    __syncthreads();

    for (int i = idx; i < nodeCount; i += blockDim.x) {
      if (i == s) {
        distance[i] = 0;
        sigma[i] = 1;
      } else {
        distance[i] = INT_MAX;
        sigma[i] = 0;
      }
      dependency[i]= 0.0;
    }
    __syncthreads();

    while (!done) {
      __syncthreads();

      if (threadIdx.x == 0){
        current_depth++;
      }
      done = true;
      __syncthreads();

      for (int i = idx; i < (2*(graph->edgeCount)); i += blockDim.x)  {
        int v = graph->edgeList1[i];
        if (distance[v] == current_depth) {    
          int w = graph->edgeList2[i];
          if (distance[w] == INT_MAX) {
            distance[w] = distance[v] + 1;
            done = false;
          }
          if (distance[w] == (distance[v] + 1)) {
            atomicAdd(&sigma[w], sigma[v]);
          }
        }
      }
      __syncthreads();
    }
    __syncthreads();

    // Reverse BFS
    while (current_depth) {
      if (idx == 0) {
        current_depth--;
      }
      __syncthreads();

      for (int i = idx; i < (2*(graph->edgeCount)); i += blockDim.x) {
        int v = graph->edgeList1[i];
        if (distance[v] == current_depth) {
          int w = graph->edgeList2[i];
          if(distance[w] == (distance[v] + 1)) {
            if (sigma[w] != 0) {
              atomicAdd(dependency + v, (sigma[v] * 1.0 / sigma[w]) * (1 + dependency[w]));
            }
          }
        }
      }
      __syncthreads();
    }

    for (int v = idx; v < nodeCount; v += blockDim.x) {
      if (v != s) {
        bwCentrality[v] += dependency[v] / 2;
      }
    }
    __syncthreads();
  }
}

float *betweennessCentrality(Graph *graph, int nodeCount, int nodeFrom, int nodeTo) {
  float *bwCentrality = new float[nodeCount]();
  float *device_bwCentrality, *dependency;
  int *sigma, *distance;

  catchCudaError(hipMalloc((void **)&device_bwCentrality, sizeof(float) * nodeCount));
  catchCudaError(hipMalloc((void **)&sigma, sizeof(int) * nodeCount));
  catchCudaError(hipMalloc((void **)&distance, sizeof(int) * nodeCount));
  catchCudaError(hipMalloc((void **)&dependency, sizeof(float) * nodeCount));
  catchCudaError(hipMemcpy(device_bwCentrality, bwCentrality, sizeof(float) * nodeCount, hipMemcpyHostToDevice));

  hipEvent_t device_start, device_end;
  catchCudaError(hipEventCreate(&device_start));
  catchCudaError(hipEventCreate(&device_end));
  catchCudaError(hipEventRecord(device_start));

  betweennessCentralityKernel<<<1, MAX_THREAD_COUNT>>>(
    graph,
    device_bwCentrality,
    nodeFrom,
    nodeTo,
    nodeCount,
    sigma,
    distance,
    dependency
  );
  hipDeviceSynchronize();
  
  // Timer
  catchCudaError(hipEventRecord(device_end));
  catchCudaError(hipEventSynchronize(device_end));
  hipEventElapsedTime(&device_time_taken, device_start, device_end);

  // Copy back and free memory
  catchCudaError(hipMemcpy(bwCentrality, device_bwCentrality, sizeof(float) * nodeCount, hipMemcpyDeviceToHost));
  catchCudaError(hipFree(device_bwCentrality));
  catchCudaError(hipFree(sigma));
  catchCudaError(hipFree(dependency));
  catchCudaError(hipFree(distance));

  return bwCentrality;
}

int main(int argc, char *argv[]) {
  if (argc < 2) {
    cout << "Usage: " << argv[0] << " <input_file> <output_file>\n";
    return 0;
  }

  freopen(argv[1], "r", stdin);

  Graph *host_graph = new Graph();
  Graph *device_graph;

  catchCudaError(hipMalloc((void **)&device_graph, sizeof(Graph)));
  host_graph->readGraph();
  host_graph->convertToCOO();

  int nodeCount = host_graph->getNodeCount();
  int edgeCount = host_graph->getEdgeCount();
  catchCudaError(hipMemcpy(device_graph, host_graph, sizeof(Graph), hipMemcpyHostToDevice));

  // Set threshold
  const long threshold_percent = strtol(argv[3], NULL, 10);
  const int threshold = (int) ((float)nodeCount * (float)threshold_percent / (float) 100);
  const int nodeFrom = threshold;
  const int nodeTo = nodeCount - 1;

  // Copy edge List to device
  int *edgeList1;
  int *edgeList2;

  // Alocate device memory and copy
  catchCudaError(hipMalloc((void **)&edgeList1, sizeof(int) * (2 * edgeCount + 1)));
  catchCudaError(hipMemcpy(edgeList1, host_graph->edgeList1, sizeof(int) * (2 * edgeCount + 1), hipMemcpyHostToDevice));

  catchCudaError(hipMalloc((void **)&edgeList2, sizeof(int) * (2 * edgeCount + 1)));
  catchCudaError(hipMemcpy(edgeList2, host_graph->edgeList2, sizeof(int) * (2 * edgeCount + 1), hipMemcpyHostToDevice));

  // Update the pointer to this, in device_graph
  catchCudaError(hipMemcpy(&(device_graph->edgeList1), &edgeList1, sizeof(int *), hipMemcpyHostToDevice));
  catchCudaError(hipMemcpy(&(device_graph->edgeList2), &edgeList2, sizeof(int *), hipMemcpyHostToDevice));

  float *bwCentrality = betweennessCentrality(device_graph, nodeCount, nodeFrom, nodeTo);

  float maxBetweenness = -1;
  for (int i = 0; i < nodeCount; i++) {
    maxBetweenness = max(maxBetweenness, bwCentrality[i]);
  }

  cout << endl;

  printf("%s, %s,", argv[1], argv[3]);
  printf("%0.2lf, %0.2lf\n", maxBetweenness, device_time_taken);

  if (argc == 3) {
    freopen(argv[2], "w", stdout);
    for (int i = 0; i < nodeCount; i++)
      cout << bwCentrality[i] << " ";
    cout << endl;
  }

  delete[] bwCentrality;
  catchCudaError(hipFree(edgeList1));
  catchCudaError(hipFree(edgeList2));
  catchCudaError(hipFree(device_graph));
}
