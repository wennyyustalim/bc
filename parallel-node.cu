#include "hip/hip_runtime.h"
#include <iostream>
#include <hip/hip_runtime.h>
#include "Graph.h"

#define MAX_THREAD_COUNT 1024
#define CEIL(a, b) ((a - 1) / b + 1)
#define catchCudaError(error) { gpuAssert((error), __FILE__, __LINE__); }

using namespace std;

float device_time_taken;

void printTime(float ms) {
  printf("%d,", (int)ms);
}

inline void gpuAssert(hipError_t error, const char *file, int line,  bool abort = false) {
  if (error != hipSuccess) {
    printf("\n====== Cuda Error Code %i ======\n %s in CUDA %s\n", error, hipGetErrorString(error));
    printf("\nIn file :%s\nOn line: %d", file, line);
    
    if(abort)
      exit(-1);
  }
}

__global__ void betweennessCentralityKernel(
  Graph *graph,
  double *bwCentrality,
  int nodeFrom,
  int nodeTo,
  int nodeCount,
  int *sigma,
  int *distance,
  double *dependency) {
    
  int idx = threadIdx.x;
  if (idx >= nodeCount)
    return;
  
  __shared__ int s;
  __shared__ int current_depth;
  __shared__ bool done;

  if(idx == 0) {
    s = nodeFrom - 1;
  }
  __syncthreads();

  while (s <= nodeTo) {
    if (idx == 0) {
      ++s;
      done = false;
      current_depth = -1;
    }
    __syncthreads();

    for (int v = idx; v < nodeCount; v += blockDim.x) {
        if (v == s) {
          distance[v] = 0;
          sigma[v] = 1;
        } else {
          distance[v] = INT_MAX;
          sigma[v] = 0;
        }
        dependency[v] = 0.0;
    }
    __syncthreads();

    while (!done) {
      if (idx == 0) {
          current_depth++;
      }
      done = true;
      __syncthreads();

      for (int v = idx; v < nodeCount; v += blockDim.x) {
        if (distance[v] == current_depth) {
          for (int r = graph->adjacencyListPointers[v]; r < graph->adjacencyListPointers[v + 1]; r++) {
            int w = graph->adjacencyList[r];
            if (distance[w] == INT_MAX) {
              distance[w] = distance[v] + 1;
              done = false;
            }
            if (distance[w] == (distance[v] + 1)) {
              atomicAdd(&sigma[w], sigma[v]);
            }
          }
        }
      }
      __syncthreads();
    }

    while(current_depth) {
      if (idx == 0) {
        current_depth--;
      }
      __syncthreads();

      for (int v = idx; v < nodeCount; v += blockDim.x) {
        if (distance[v] == current_depth) {
          for (int r = graph->adjacencyListPointers[v]; r < graph->adjacencyListPointers[v + 1]; r++) {
            int w = graph->adjacencyList[r];
            if (distance[w] == (distance[v] + 1)) {
              if (sigma[w] != 0)
                dependency[v] += (sigma[v] * 1.0 / sigma[w]) * (1 + dependency[w]);
            }
          }

          if (v != s) {
            bwCentrality[v] += dependency[v] / 2;
          }
        }
      }
      __syncthreads();
    }
  }
}

double *betweennessCentrality(Graph *graph, int nodeCount, int nodeFrom, int nodeTo) {
  double *bwCentrality = new double[nodeCount]();
  double *device_bwCentrality, *dependency;
  int *sigma, *distance;

  catchCudaError(hipMalloc((void **)&device_bwCentrality, sizeof(double) * nodeCount));
  catchCudaError(hipMalloc((void **)&sigma, sizeof(int) * nodeCount));
  catchCudaError(hipMalloc((void **)&distance, sizeof(int) * nodeCount));
  catchCudaError(hipMalloc((void **)&dependency, sizeof(double) * nodeCount));
  catchCudaError(hipMemcpy(device_bwCentrality, bwCentrality, sizeof(double) * nodeCount, hipMemcpyHostToDevice));

  // Timer
  hipEvent_t device_start, device_end;
  catchCudaError(hipEventCreate(&device_start));
  catchCudaError(hipEventCreate(&device_end));
  catchCudaError(hipEventRecord(device_start));

  betweennessCentralityKernel<<<1, MAX_THREAD_COUNT>>>(
    graph,
    device_bwCentrality,
    nodeFrom,
    nodeTo,
    nodeCount,
    sigma,
    distance,
    dependency
  );
  hipDeviceSynchronize();
  cout << endl;

  // Timer
  catchCudaError(hipEventRecord(device_end));
  catchCudaError(hipEventSynchronize(device_end));
  hipEventElapsedTime(&device_time_taken, device_start, device_end);

  // Copy back and free memory
  catchCudaError(hipMemcpy(bwCentrality, device_bwCentrality, sizeof(double) * nodeCount, hipMemcpyDeviceToHost));
  catchCudaError(hipFree(device_bwCentrality));
  catchCudaError(hipFree(sigma));
  catchCudaError(hipFree(dependency));
  catchCudaError(hipFree(distance));
  return bwCentrality;
}

int main(int argc, char *argv[]) {
  if (argc < 2) {
    cout << "Usage: " << argv[0] << " <input_file> <output_file>\n";
    return 0;
  }

  freopen(argv[1], "r", stdin);

  Graph *host_graph = new Graph();
  Graph *device_graph;

  catchCudaError(hipMalloc((void **)&device_graph, sizeof(Graph)));
  host_graph->readGraph();

  int nodeCount = host_graph->getNodeCount();
  int edgeCount = host_graph->getEdgeCount();
  catchCudaError(hipMemcpy(device_graph, host_graph, sizeof(Graph), hipMemcpyHostToDevice));

  // Set threshold
  const long threshold_percent = strtol(argv[3], NULL, 10);
  const int threshold = (int) ((float)nodeCount * (float)threshold_percent / (float) 100);
  const int nodeFrom = threshold;
  const int nodeTo = nodeCount - 1;

  int *adjacencyList;
  catchCudaError(hipMalloc((void **)&adjacencyList, sizeof(int) * (2 * edgeCount + 1)));
  catchCudaError(hipMemcpy(adjacencyList, host_graph->adjacencyList, sizeof(int) * (2 * edgeCount + 1), hipMemcpyHostToDevice));
  catchCudaError(hipMemcpy(&(device_graph->adjacencyList), &adjacencyList, sizeof(int *), hipMemcpyHostToDevice));

  int *adjacencyListPointers;
  catchCudaError(hipMalloc((void **)&adjacencyListPointers, sizeof(int) * (nodeCount + 1)));
  catchCudaError(hipMemcpy(adjacencyListPointers, host_graph->adjacencyListPointers, sizeof(int) * (nodeCount + 1), hipMemcpyHostToDevice));
  catchCudaError(hipMemcpy(&(device_graph->adjacencyListPointers), &adjacencyListPointers, sizeof(int *), hipMemcpyHostToDevice));

  double *bwCentrality = betweennessCentrality(device_graph, nodeCount, nodeFrom, nodeTo);

  double maxBetweenness = -1;
  for (int i = 0; i < nodeCount; i++) {
    maxBetweenness = max(maxBetweenness, bwCentrality[i]);
  }

  printf("%s, %03d, ", argv[1], atoi(argv[3]));
  // printf("%0.2lf, ", maxBetweenness);
  printf("%0.2lf\n", device_time_taken);

  if (argc == 3) {
    freopen(argv[2], "w", stdout);
    for (int i = 0; i < nodeCount; i++)
      cout << bwCentrality[i] << " ";
    cout << endl;
  }

  // Free all memory
  delete[] bwCentrality;
  catchCudaError(hipFree(adjacencyList));
  catchCudaError(hipFree(adjacencyListPointers));
  catchCudaError(hipFree(device_graph));
}
